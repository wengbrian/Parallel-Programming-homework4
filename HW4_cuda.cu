#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

const int INF = 1000000000;
const int V = 20010;
void input(char *inFileName);
void output(char *outFileName);
int *d_ptr;
size_t pitch;
int debug = 0;

void copyTo();
void copyBack();
void block_FW(int B);
int ceil(int a, int b);
__global__ void cal(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height, int n, int *d_ptr, size_t pitch, int num_block, int debug);

int n, m;	// Number of vertices, edges
static int Dist[V][V];

int main(int argc, char* argv[])
{
	input(argv[1]);
	int B = atoi(argv[3]);
    copyTo();
	block_FW(B);
    copyBack();

	output(argv[2]);

	return 0;
}

void input(char *inFileName)
{
	FILE *infile = fopen(inFileName, "r");
	fscanf(infile, "%d %d", &n, &m);

	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			if (i == j)	Dist[i][j] = 0;
			else		Dist[i][j] = INF;
		}
	}

	while (--m >= 0) {
		int a, b, v;
		fscanf(infile, "%d %d %d", &a, &b, &v);
		Dist[a][b] = v;
	}
    fclose(infile);
}

void output(char *outFileName)
{
	FILE *outfile = fopen(outFileName, "w");
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			// if (Dist[i][j] >= INF)	fprintf(outfile, "INF ");
			// else					fprintf(outfile, "%d ", Dist[i][j]);
            if (Dist[i][j] >= INF)
                Dist[i][j] = INF;
		}
		fwrite(Dist[i], sizeof(int), n, outfile);
	}
    fclose(outfile);
}

int ceil(int a, int b)
{
	return (a + b -1)/b;
}

void copyTo(){
    if(debug){
        printf("\nhost:\n");
        for(int i = 0; i < 5; i++){
            int *row =  Dist[i];
            for(int j = 0; j < 5; j++){
                printf("%d ", row[j]);
            }
            printf("\n");
        }
    }
    hipMallocPitch(&d_ptr, &pitch, n * sizeof(int), n);
    hipMemcpy2D(d_ptr, pitch, &Dist, V * sizeof(int), n * sizeof(int), n, hipMemcpyHostToDevice);
}

void copyBack(){
    hipMemcpy2D(&Dist, V * sizeof(int), d_ptr, pitch, n * sizeof(int), n, hipMemcpyDeviceToHost);
    if(debug){
        printf("\nhost:\n");
        for(int i = 0; i < 5; i++){
            int *row =  Dist[i];
            for(int j = 0; j < 5; j++){
                printf("%d ", row[j]);
            }
            printf("\n");
        }
    }
}
__global__
void test(int *d_ptr, size_t pitch, int n){
    printf("\ndevice:\n");
    for(int i = 0; i < n; i++){
        int *row = (int*) ((char*)d_ptr + i * pitch);
        for(int j = 0; j < n; j++){
            printf("%d ", row[j]);
        }
        printf("\n");
    }
}

void block_FW(int B)
{
	int round = ceil(n, B);
    int size = 3*1024*sizeof(int);
    dim3 block(32,32);
    int num_block = ceil(B,32);
	for (int r = 0; r < round; ++r) {
        if(debug)
            printf("%d %d\n", r, round);

		/* Phase 1*/
        int num_block_width = ceil(B,32);
        int num_block_height = ceil(B,32);
        dim3 grid(num_block_width, num_block_height);
		cal<<<grid,block,size>>>(B,	r,	r,	r,	1,	1, n, d_ptr, pitch, num_block, debug);
        hipDeviceSynchronize();

		/* Phase 2*/
        if(r > 0){
            int num_block_width = ceil(B,32);
            int num_block_height = ceil(B*r,32);
            dim3 grid(num_block_width, num_block_height);
            cal<<<grid,block,size>>>(B, r, r, 0, 1, r, n, d_ptr, pitch, num_block, debug); // up
        }
        if(round-r-1 > 0){
            int num_block_width = ceil(B,32);
            int num_block_height = ceil(B*(round-r-1),32);
            dim3 grid(num_block_width, num_block_height);
		    cal<<<grid,block,size>>>(B, r, r, r+1, 1, round-r-1, n, d_ptr, pitch, num_block, debug); // down
        }
        if(r > 0){
            int num_block_width = ceil(B*r,32);
            int num_block_height = ceil(B,32);
            dim3 grid(num_block_width, num_block_height);
		    cal<<<grid,block,size>>>(B, r, 0, r, r, 1, n, d_ptr, pitch, num_block, debug); // left
        }
        if(round-r-1 > 0){
            int num_block_width = ceil(B*(round-r-1),32);
            int num_block_height = ceil(B,32);
            dim3 grid(num_block_width, num_block_height);
		    cal<<<grid,block,size>>>(B, r, r+1, r, round-r-1, 1, n, d_ptr, pitch, num_block, debug); // right
        }
        hipDeviceSynchronize();

		/* Phase 3*/
        if(r > 0){
            int num_block_width = ceil(B*r,32);
            int num_block_height = ceil(B*r,32);
            dim3 grid(num_block_width, num_block_height);
            cal<<<grid,block,size>>>(B, r, 0, 0, r, r, n, d_ptr, pitch, num_block, debug); // left-up
        }
        if((round-r-1 > 0) && (r > 0)){
            int num_block_width = ceil(B*r,32);
            int num_block_height = ceil(B*(round-r-1),32);
            dim3 grid(num_block_width, num_block_height);
		    cal<<<grid,block,size>>>(B, r, 0, r+1, r, round-r-1, n, d_ptr, pitch, num_block, debug); // left-down
        }
        if((round-r-1 > 0) && (r > 0)){
            int num_block_width = ceil(B*(round-r-1),32);
            int num_block_height = ceil(B*r,32);
            dim3 grid(num_block_width, num_block_height);
		    cal<<<grid,block,size>>>(B, r, r +1, 0, round-r-1, r, n, d_ptr, pitch, num_block, debug); // right-up
        }
        if(round-r-1 > 0){
            int num_block_width = ceil(B*(round-r-1),32);
            int num_block_height = ceil(B*(round-r-1),32);
            dim3 grid(num_block_width, num_block_height);
		    cal<<<grid,block,size>>>(B, r, r+1, r+1, round-r-1, round-r-1, n, d_ptr, pitch, num_block, debug); // right-down
        }
        hipDeviceSynchronize();
        
	}
    if(debug)
        test<<<1,1>>>(d_ptr, pitch, n);
}

__global__
void cal(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height, int n, int *d_ptr, size_t pitch, int num_block, int debug)
{
    int size = B < 32 ? B : 32;
    // shared memory
    __shared__ int mem[3072];
    int *dist = mem;
    int *rowBlock = dist + 1024;
    int *colBlock = rowBlock + 1024;

    // num block for each part
    //int num_block_width = gridDim.x;
    //int num_block_height = gridDim.y;

    int idx = threadIdx.y * blockDim.x + threadIdx.x; // internal index, used by dist
    int tidx = block_start_x * B + 
               blockIdx.x * blockDim.x + 
               threadIdx.x; // external index, used by d_ptr
    int tidy = block_start_y * B +
               blockIdx.y * blockDim.y + 
               threadIdx.y; // external index, used by d_ptr
    int b_x = blockIdx.x * blockDim.x + threadIdx.x;
    int b_y = blockIdx.y * blockDim.y + threadIdx.y;
    // copy memory from global memory to shared memory
    // copy current block
    if ((tidx < n) && (tidy < n) && (b_x < block_width*B) && (b_y < block_height*B)){
        int *row = (int *)((char*)d_ptr + tidy * pitch);
        dist[idx] = row[tidx]; 
    }

    for (int i = 0; i < num_block; i++){
        // copy needed row block
        int idx_x = Round*B + threadIdx.x + i*32; // k_x
        //if((idx_x < (Round+1)*B) && (idx_x < n) && (tidy < n)){
        if(true){
            int *row = (int *)((char*)d_ptr + tidy * pitch);
            rowBlock[idx] = row[idx_x];
            //printf("copy d[%d][%d] = %d\n", tidx, tidy, row[idx_x]);
        }

        // copy needed col block
        int idx_y = Round*B + threadIdx.y + i*32;
        //if((idx_y < (Round+1)*B) && (idx_y < n) && (tidx < n)){
        if(true){
            int *row = (int *)((char*)d_ptr + idx_y * pitch);
            colBlock[idx] = row[tidx];
            //printf("copy d[%d][%d] = %d\n", tidx, tidy, row[idx_x]);
        }
        __syncthreads();

        // current k = Round*B ~ Round*B+size-1
        for(int k = 0; (k < size) && (Round*B+k < n); k++){
            //if ((tidx < n) && (tidy < n) && (b_x < block_width*B) && (b_y < block_height*B)){
            if ((tidx < n) && (tidy < n) && (b_x < block_width*B) && (b_y < block_height*B)){
                int ik = rowBlock[threadIdx.y*32+k];
                int kj = colBlock[k*32+threadIdx.x];
                if(debug)printf("[%d,%d][k=%d, ex_tidx=%d, ex_tidy=%d, in_tidx=%d, in_tidy=%d, idx=%d] if %d + %d < %d\n", blockIdx.x, blockIdx.y, Round*B+k, tidx, tidy, threadIdx.x, threadIdx.y, idx, ik, kj, dist[idx]);
                if (ik + kj < dist[idx]) {
                    dist[idx] = ik + kj;
                }
            }
            __syncthreads();
        }
    }

    // copy shared memory to global memory
    if ((tidx < n) && (tidy < n) && (b_x < block_width*B) && (b_y < block_height*B)){
        int *row = (int *)((char*)d_ptr + tidy * pitch);
        row[tidx] = dist[idx]; 
    }
}


